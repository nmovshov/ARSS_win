#include "hip/hip_runtime.h"
#include "cuShared.h"
#include <math.h>

#define TILE_WIDTH 64
#define EPS2 0.00125f

// global device variables and functions
float4 *g_devicePositionsArr=NULL;
float4 *g_hostPositionsArr=NULL;
float3 *g_deviceForcesArr=NULL;
float3 *g_hostForcesArr=NULL;

__global__ void GravityKernel(unsigned int bodies, float4 *bodyDescripts, float3 *forcesToUpodate);
__global__ void calculate_forces(float4 *bodyDescripts, float3 *forcesToUpdate, unsigned int N);
__device__ float3 bodyBodyInteraction(float4 bi, float4 bj, float3 ai);

bool AllocateCUDAGlobals(int nbThings)
{
	// Not just allocate. This is a convenient place to create CUDA debugging information and look into capabilities of physical devices
	hipDeviceProp_t gpuProps;
	int devToUse=1;
	hipSetDevice(devToUse);
	hipGetDeviceProperties(&gpuProps,devToUse);

	printf("Device name=%s\n",gpuProps.name);
	printf("Compute capability=%d.%d\n",gpuProps.major,gpuProps.minor);
	printf("Total global memory=%d kB\n",gpuProps.totalGlobalMem/1024);
	printf("Total shared memory per block=%d kB\n",gpuProps.sharedMemPerBlock/1024);
	printf("Max Grid size=%d x %d x %d\n",gpuProps.maxGridSize[0],gpuProps.maxGridSize[1],gpuProps.maxGridSize[2]);
	printf("Max threads per block=%d x %d x %d\n",gpuProps.maxThreadsDim[0],gpuProps.maxThreadsDim[1],gpuProps.maxThreadsDim[2]);
	printf("Max threads per block=%d\n",gpuProps.maxThreadsPerBlock);
	printf("Multi-processors count=%d\n",gpuProps.multiProcessorCount);
	printf("Kernel execution may time out=%d\n",gpuProps.kernelExecTimeoutEnabled);
	printf("Compute mode=%d\n",gpuProps.computeMode);
	printf("ECC enabled=%d\n",gpuProps.ECCEnabled);

	// allocate memory for device and host arrays of actor positions and forces
	hipError_t err;
	size_t nbytes4=nbThings * sizeof(float) * 4;
	size_t nbytes3=nbThings * sizeof(float) * 3;
	err = hipMalloc((void **) &g_devicePositionsArr,nbytes4);
	if (err != hipSuccess) {
		printf("Error in hipMalloc()...%s\a\n",hipGetErrorString(err));
		exit(1);
	}
	err = hipMalloc((void **) &g_deviceForcesArr,nbytes3);
	if (err != hipSuccess) {
		printf("Error in hipMalloc()...%s\a\n",hipGetErrorString(err));
		exit(1);
	}

	g_hostPositionsArr = (float4 *) malloc(nbytes4);
	g_hostForcesArr = (float3 *) malloc(nbytes3);

	//err = hipFuncSetCacheConfig(reinterpret_cast<const void*>(GravityKernel),hipFuncCachePreferL1);
	//err = hipFuncSetCacheConfig(reinterpret_cast<const void*>(calculate_forces),hipFuncCachePreferShared);

	return true;
}

void ReleaseCUDA()
{
	hipError_t err;
	err=hipFree(g_devicePositionsArr);
	err=hipFree(g_deviceForcesArr);
	free(g_hostPositionsArr);
	free(g_hostForcesArr);
}

void GravitateOnDevice(unsigned int bodies, unsigned int mode)
	// mode controls the async behavior. mode=1 will set up and launch the gravity kernel and then return. mode=2 will complete the gravity
	// kernel and retrieve the forces. mode=3 will do everything.
{
	hipError_t err;
	size_t nbytes3 = bodies * sizeof(float) * 3;
	size_t nbytes4 = bodies * sizeof(float) * 4;

	if (mode==1 || mode==3)
	{
		// copy the body descriptions to device global memory
		err = hipMemcpy(g_devicePositionsArr,g_hostPositionsArr,nbytes4,hipMemcpyHostToDevice);

		if (err!=hipSuccess){
			printf("cuda memcpy failed!\n");
			exit(1);
		}

		// reset device forces
		err = hipMemset(g_deviceForcesArr,0,nbytes3);

		// launch a device force calculation
		int nt=(TILE_WIDTH<bodies)?TILE_WIDTH:bodies;
		int nb = (int) ceil((float)bodies/(float)nt); // recommended to use a good multiple of 32 for bodies
		dim3 grid(nb,1,1);
		dim3 block(nt,1,1);


		calculate_forces<<<grid,block>>>(g_devicePositionsArr,g_deviceForcesArr,bodies);	
		//GravityKernel<<<grid,block>>>(bodies, g_devicePositionsArr, g_deviceForcesArr);
		err=hipGetLastError();
		if (err!=hipSuccess)
		{
			printf("kernel launch failed!\a\n");
			exit(1);
		}
	}

	if (mode==2 || mode==3)
	{
		// copy updated forces from device to host
		err = hipDeviceSynchronize();
		err = hipMemcpy(g_hostForcesArr,g_deviceForcesArr,nbytes3,hipMemcpyDeviceToHost);
		if (err!=hipSuccess){
			printf("cuda memcpy failed!\n");
			exit(1);
		}
	}

	return;
}

__global__ void GravityKernel(unsigned int bodies, float4 *bodyDescripts, float3 *forcesToUpdate)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid<bodies)
	{
		float4 rp = bodyDescripts[tid];
		float4 r;
		float distSqr, invDist, invDistCube, s;

		for (int k=0; k<bodies; k++)
		{
			if (k==tid) continue;
			r = bodyDescripts[k];
			r.x -= rp.x;
			r.y -= rp.y;
			r.z -= rp.z;

			distSqr = r.x*r.x + r.y*r.y + r.z*r.z;
			invDist = 1.0f/sqrtf(distSqr);
			invDistCube = invDist*invDist*invDist;

			s = r.w * rp.w * invDistCube; // universal big G added outside

			forcesToUpdate[tid].x += r.x * s;
			forcesToUpdate[tid].y += r.y * s;
			forcesToUpdate[tid].z += r.z * s;
		}
	}
	return;
}

__device__ float3 bodyBodyInteraction(float4 bi, float4 bj, float3 ai)
{
	float3 r;
	// r_ij [3 flops]
	r.x = bj.x - bi.x;
	r.y = bj.y - bi.y;
	r.z = bj.z - bi.z;

	// distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
	float distSqr = r.x*r.x + r.y*r.y + r.z*r.z + EPS2;

	// invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
	float distSixth = distSqr * distSqr * distSqr;
	float invDistCube = 1.0f/ sqrtf (distSixth);
	
	// s = m_j * invDistCube [1 FLOP]
	float s = bi.w * bj.w * invDistCube; // universal big G added outside
	
	// a_i =  a_i + s * r_ij [6 FLOPS]
	ai.x += r.x * s;
	ai.y += r.y * s;
	ai.z += r.z * s;
	
	return ai;
}

__global__ void calculate_forces(float4 *bodyDescripts, float3 *forcesToUpdate, unsigned int N)
{
	__shared__ float4 shPosition[TILE_WIDTH];

	float4 myPosition;
	int j, tile;
	float3 acc = {0.0f, 0.0f, 0.0f};
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;

	myPosition=bodyDescripts[gtid];

	for (j = 0, tile = 0; j < N; j+=TILE_WIDTH, tile++) {
		int idx = tile * blockDim.x + threadIdx.x;
		shPosition[threadIdx.x] = bodyDescripts[idx];
		__syncthreads();
		for (int k=0; k<TILE_WIDTH; k++)
		{
			acc=bodyBodyInteraction(myPosition,shPosition[k],acc);
		}
		__syncthreads();
	}
	// Save the result in global memory for the integration step.
	//float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
	//globalA[gtid] = acc4;
	forcesToUpdate[gtid]=acc;
}